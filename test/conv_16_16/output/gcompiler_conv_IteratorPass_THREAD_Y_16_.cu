
#include <hip/hip_runtime.h>
#define COALESCED_NUM 16
#define blockDimX 256
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 16
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_C 2048
#define WIDTH_B 16
#define WIDTH_A (2048+16)
__global__ void conv(float * A, float * B, float * C, int width, int height, int w, int h)
{
	__shared__ float shared_1[16][17];
	__shared__ float shared_0[272];
	int j;
	float sum_0 = 0;
	float sum_1 = 0;
	float sum_2 = 0;
	float sum_3 = 0;
	float sum_4 = 0;
	float sum_5 = 0;
	float sum_6 = 0;
	float sum_7 = 0;
	float sum_8 = 0;
	float sum_9 = 0;
	float sum_10 = 0;
	float sum_11 = 0;
	float sum_12 = 0;
	float sum_13 = 0;
	float sum_14 = 0;
	float sum_15 = 0;
	int it_2;
	for (j=0; j<(h-15); j=(j+1))
	{
		int it_2;
		if ((tidx<16))
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*j))+h), (idx+(( - 1)*0)));
		}
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*j))+h), ((idx+(( - 1)*0))+16));
		__syncthreads();
		if ((tidx<16))
		{
			shared_1[(tidx+0)][0]=B((j+0), (0+tidx));
			shared_1[(tidx+0)][1]=B((j+1), (0+tidx));
			shared_1[(tidx+0)][2]=B((j+2), (0+tidx));
			shared_1[(tidx+0)][3]=B((j+3), (0+tidx));
			shared_1[(tidx+0)][4]=B((j+4), (0+tidx));
			shared_1[(tidx+0)][5]=B((j+5), (0+tidx));
			shared_1[(tidx+0)][6]=B((j+6), (0+tidx));
			shared_1[(tidx+0)][7]=B((j+7), (0+tidx));
			shared_1[(tidx+0)][8]=B((j+8), (0+tidx));
			shared_1[(tidx+0)][9]=B((j+9), (0+tidx));
			shared_1[(tidx+0)][10]=B((j+10), (0+tidx));
			shared_1[(tidx+0)][11]=B((j+11), (0+tidx));
			shared_1[(tidx+0)][12]=B((j+12), (0+tidx));
			shared_1[(tidx+0)][13]=B((j+13), (0+tidx));
			shared_1[(tidx+0)][14]=B((j+14), (0+tidx));
			shared_1[(tidx+0)][15]=B((j+15), (0+tidx));
		}
		__syncthreads();
		#pragma unroll 
		for (it_2=0; it_2<16; it_2=(it_2+1))
		{
			float a;
			float b_0;
			float b_1;
			float b_2;
			float b_3;
			float b_4;
			float b_5;
			float b_6;
			float b_7;
			float b_8;
			float b_9;
			float b_10;
			float b_11;
			float b_12;
			float b_13;
			float b_14;
			float b_15;
			a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
			b_0=shared_1[it_2][0];
			b_1=shared_1[it_2][1];
			b_2=shared_1[it_2][2];
			b_3=shared_1[it_2][3];
			b_4=shared_1[it_2][4];
			b_5=shared_1[it_2][5];
			b_6=shared_1[it_2][6];
			b_7=shared_1[it_2][7];
			b_8=shared_1[it_2][8];
			b_9=shared_1[it_2][9];
			b_10=shared_1[it_2][10];
			b_11=shared_1[it_2][11];
			b_12=shared_1[it_2][12];
			b_13=shared_1[it_2][13];
			b_14=shared_1[it_2][14];
			b_15=shared_1[it_2][15];
			sum_0+=(a*b_0);
			sum_1+=(a*b_1);
			sum_2+=(a*b_2);
			sum_3+=(a*b_3);
			sum_4+=(a*b_4);
			sum_5+=(a*b_5);
			sum_6+=(a*b_6);
			sum_7+=(a*b_7);
			sum_8+=(a*b_8);
			sum_9+=(a*b_9);
			sum_10+=(a*b_10);
			sum_11+=(a*b_11);
			sum_12+=(a*b_12);
			sum_13+=(a*b_13);
			sum_14+=(a*b_14);
			sum_15+=(a*b_15);
		}
		__syncthreads();
		__syncthreads();
	}
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-1)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-1)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		sum_0+=(a*b_0);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-2)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-2)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-3)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-3)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-4)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-4)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-5)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-5)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-6)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-6)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-7)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-7)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-8)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-8)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-9)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-9)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-10)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-10)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-11)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-11)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-11), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-12)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-12)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-12), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-11), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-13)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-13)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-13), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-12), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-11), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-14)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-14)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-14), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-13), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-12), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-11), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(h-15)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(h-15)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-15), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-14), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-13), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-12), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-11), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-10), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-9), (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B((h-8), (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
	}
	C(((idy*16)+0), idx)=sum_0;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-1)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-1)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][1]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(9, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(10, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(11, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(12, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(13, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(14, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+1), idx)=sum_1;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-2)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-2)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][2]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(9, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(10, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(11, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(12, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(13, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+2), idx)=sum_2;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-3)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-3)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][3]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(9, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(10, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(11, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(12, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+3), idx)=sum_3;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-4)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-4)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][4]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(9, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(10, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(11, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+4), idx)=sum_4;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-5)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-5)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][5]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(9, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(10, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_5;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+5), idx)=sum_5;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-6)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-6)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][6]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(8, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(9, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_6;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+6), idx)=sum_6;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-7)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-7)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][7]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][8]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(7, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(8, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_7;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_7=shared_1[it_2][7];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_7+=(a*b_7);
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+7), idx)=sum_7;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-8)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-8)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][8]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][9]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(6, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(7, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_8;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_8=shared_1[it_2][8];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_8+=(a*b_8);
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+8), idx)=sum_8;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-9)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-9)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][9]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][10]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(6, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_9;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_9=shared_1[it_2][9];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_9+=(a*b_9);
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+9), idx)=sum_9;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-10)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-10)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][10]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][11]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(5, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_10;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_10=shared_1[it_2][10];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_10+=(a*b_10);
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+10), idx)=sum_10;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-11)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-11)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][11]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][12]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(4, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_11;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_11=shared_1[it_2][11];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_11+=(a*b_11);
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+11), idx)=sum_11;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-12)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-12)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][12]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][13]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(3, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_12;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_12=shared_1[it_2][12];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_12+=(a*b_12);
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+12), idx)=sum_12;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-13)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-13)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][13]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][14]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(2, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_13;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_13=shared_1[it_2][13];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_13+=(a*b_13);
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+13), idx)=sum_13;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-14)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-14)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][14]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][15]=B(1, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_14;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_14=shared_1[it_2][14];
		b_15=shared_1[it_2][15];
		sum_14+=(a*b_14);
		sum_15+=(a*b_15);
	}
	C(((idy*16)+14), idx)=sum_14;
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*16)+(( - 1)*(0-15)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*16)+(( - 1)*(0-15)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][15]=B(0, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll 
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_15;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_15=shared_1[it_2][15];
		sum_15+=(a*b_15);
	}
	C(((idy*16)+15), idx)=sum_15;
	__syncthreads();
	__syncthreads();
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
}
