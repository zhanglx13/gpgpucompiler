
#include <hip/hip_runtime.h>
#define COALESCED_NUM 16
#define blockDimX 128
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_C 2048
#define WIDTH_B 2048
#define WIDTH_A 2048
__global__ void matmul(float * A, float * B, float * C, int width, int height)
{
	__shared__ float shared_0[16];
	int i;
	float sum;
	sum=0;
	for (i=0; i<width; i=(i+16))
	{
		int it_1;
		if ((tidx<16))
		{
			shared_0[(tidx+0)]=A(idy, (i+tidx));
		}
		__syncthreads();
		#pragma unroll 
		for (it_1=0; it_1<16; it_1=(it_1+1))
		{
			float a;
			float b;
			a=shared_0[it_1];
			b=B((it_1+i), idx);
			sum+=(a*b);
		}
		__syncthreads();
	}
	{
		C(idy, idx)=sum;
	}
}
