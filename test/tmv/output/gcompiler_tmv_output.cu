
#include <hip/hip_runtime.h>
#define COALESCED_NUM 32
#define blockDimX 512
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimY 1
#define WIDTH_A 2048
#define A(y,x) A[(y)*WIDTH_A+(x)]
__global__ void tmv(float * A, float * B, float * C, int width)
{
	__shared__ float shared_0[32];
	int i;
	float sum;
	i=0;
	sum=0;
	float tmp_0;
	if ((tidx<32))
	{
		tmp_0=B[(0+tidx)];
	}
	for (i=0; i<width; i=(i+32))
	{
		int it_1;
		if ((tidx<32))
		{
			shared_0[(tidx+0)]=tmp_0;
		}
		__syncthreads();
		#pragma unroll 
		for (it_1=0; it_1<32; it_1=(it_1+1))
		{
			float a;
			float b;
			a=A((it_1+i), idx);
			b=shared_0[it_1];
			sum+=(a*b);
		}
		if ((tidx<32))
		{
			if ((i<(width-32)))
			{
				tmp_0=B[((i+32)+tidx)];
			}
		}
		__syncthreads();
	}
	{
		C[idx]=sum;
	}
}
