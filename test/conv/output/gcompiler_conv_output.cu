
#include <hip/hip_runtime.h>
#define COALESCED_NUM 16
#define blockDimX 256
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_C 2048
#define WIDTH_B 16
#define WIDTH_A (2048+16)
__global__ void conv(float * A, float * B, float * C, int width, int height, int w, int h)
{
	__shared__ float shared_1[16];
	__shared__ float shared_0[272];
	int i;
	int j;
	float sum = 0;
	for (j=0; j<h; j=(j+1))
	{
		for (i=0; i<w; i=(i+16))
		{
			int it_2;
			if ((tidx<16))
			{
				shared_0[(tidx+0)]=A(((idy+(( - 1)*j))+h), (((idx+(( - 1)*i))+w)+( - 16)));
			}
			shared_0[(tidx+16)]=A(((idy+(( - 1)*j))+h), ((idx+(( - 1)*i))+w));
			__syncthreads();
			if ((tidx<16))
			{
				shared_1[(tidx+0)]=B(j, ((i+0)+tidx));
			}
			__syncthreads();
			#pragma unroll 
			for (it_2=0; it_2<16; it_2=(it_2+1))
			{
				float a;
				float b;
				a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
				b=shared_1[it_2];
				sum+=(a*b);
			}
			__syncthreads();
			__syncthreads();
		}
	}
	{
		C(idy, idx)=sum;
	}
}
