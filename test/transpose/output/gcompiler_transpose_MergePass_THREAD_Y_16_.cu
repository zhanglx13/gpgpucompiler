#include "hip/hip_runtime.h"
#define COALESCED_NUM 32
#define blockDimX 32
#define blockDimY 4
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 16
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define A(y,x) A[(y)*WIDTH_A+(x)]
__global__ void transpose(float * A, float * C, int width)
{
	__shared__ float shared_0[32][33];
	float sum_0 = 0;
	float sum_1 = 0;
	float sum_2 = 0;
	float sum_3 = 0;
	float sum_4 = 0;
	float sum_5 = 0;
	float sum_6 = 0;
	float sum_7 = 0;
	float sum_8 = 0;
	float sum_9 = 0;
	float sum_10 = 0;
	float sum_11 = 0;
	float sum_12 = 0;
	float sum_13 = 0;
	float sum_14 = 0;
	float sum_15 = 0;
	int it_2;
	#pragma unroll 
	for (it_2=0; it_2<32; it_2=(it_2+4))
	{
		shared_0[(it_2+(tidy*1))][tidx]=A(((idx+(( - 1)*tidx))+(it_2+(tidy*1))), (coalesced_idy+tidx));
	}
	__syncthreads();
	sum_0=shared_0[tidx][((((bidy*64)+tidy)+0)+(( - 1)*coalesced_idy))];
	sum_1=shared_0[tidx][((((bidy*64)+tidy)+4)+(( - 1)*coalesced_idy))];
	sum_2=shared_0[tidx][((((bidy*64)+tidy)+8)+(( - 1)*coalesced_idy))];
	sum_3=shared_0[tidx][((((bidy*64)+tidy)+12)+(( - 1)*coalesced_idy))];
	sum_4=shared_0[tidx][((((bidy*64)+tidy)+16)+(( - 1)*coalesced_idy))];
	sum_5=shared_0[tidx][((((bidy*64)+tidy)+20)+(( - 1)*coalesced_idy))];
	sum_6=shared_0[tidx][((((bidy*64)+tidy)+24)+(( - 1)*coalesced_idy))];
	sum_7=shared_0[tidx][((((bidy*64)+tidy)+28)+(( - 1)*coalesced_idy))];
	sum_8=shared_0[tidx][((((bidy*64)+tidy)+32)+(( - 1)*coalesced_idy))];
	sum_9=shared_0[tidx][((((bidy*64)+tidy)+36)+(( - 1)*coalesced_idy))];
	sum_10=shared_0[tidx][((((bidy*64)+tidy)+40)+(( - 1)*coalesced_idy))];
	sum_11=shared_0[tidx][((((bidy*64)+tidy)+44)+(( - 1)*coalesced_idy))];
	sum_12=shared_0[tidx][((((bidy*64)+tidy)+48)+(( - 1)*coalesced_idy))];
	sum_13=shared_0[tidx][((((bidy*64)+tidy)+52)+(( - 1)*coalesced_idy))];
	sum_14=shared_0[tidx][((((bidy*64)+tidy)+56)+(( - 1)*coalesced_idy))];
	sum_15=shared_0[tidx][((((bidy*64)+tidy)+60)+(( - 1)*coalesced_idy))];
	__syncthreads();
	__syncthreads();
	{
		C((((bidy*64)+tidy)+0), idx)=sum_0;
	}
	{
		C((((bidy*64)+tidy)+4), idx)=sum_1;
	}
	{
		C((((bidy*64)+tidy)+8), idx)=sum_2;
	}
	{
		C((((bidy*64)+tidy)+12), idx)=sum_3;
	}
	{
		C((((bidy*64)+tidy)+16), idx)=sum_4;
	}
	{
		C((((bidy*64)+tidy)+20), idx)=sum_5;
	}
	{
		C((((bidy*64)+tidy)+24), idx)=sum_6;
	}
	{
		C((((bidy*64)+tidy)+28), idx)=sum_7;
	}
	{
		C((((bidy*64)+tidy)+32), idx)=sum_8;
	}
	{
		C((((bidy*64)+tidy)+36), idx)=sum_9;
	}
	{
		C((((bidy*64)+tidy)+40), idx)=sum_10;
	}
	{
		C((((bidy*64)+tidy)+44), idx)=sum_11;
	}
	{
		C((((bidy*64)+tidy)+48), idx)=sum_12;
	}
	{
		C((((bidy*64)+tidy)+52), idx)=sum_13;
	}
	{
		C((((bidy*64)+tidy)+56), idx)=sum_14;
	}
	{
		C((((bidy*64)+tidy)+60), idx)=sum_15;
	}
}
