#include "hip/hip_runtime.h"
#define COALESCED_NUM 32
#define blockDimX 32
#define blockDimY 4
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 8
#define coalesced_idy (nbidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define A(y,x) A[(y)*WIDTH_A+(x)]
__global__ void transpose(float * A, float * C, int width)
{
	int tmp_0;
	int nbidy;
	int nbidx;
	if ((gridDimX==gridDimY))
	{
		nbidy=bidx;
		nbidx=((bidx+bidy)%gridDimX);
	}else
	{
		tmp_0=(bidx+(gridDimX*bidy));
		nbidy=(tmp_0%gridDimY);
		nbidx=(((tmp_0/gridDimY)+nbidy)%gridDimX);
	}
	__shared__ float shared_0[32][33];
	float sum_0 = 0;
	float sum_1 = 0;
	float sum_2 = 0;
	float sum_3 = 0;
	float sum_4 = 0;
	float sum_5 = 0;
	float sum_6 = 0;
	float sum_7 = 0;
	int it_2;
	#pragma unroll 
	for (it_2=0; it_2<32; it_2=(it_2+4))
	{
		shared_0[(it_2+(tidy*1))][tidx]=A((((tidx+(blockDimX*nbidx))+(( - 1)*tidx))+(it_2+(tidy*1))), (coalesced_idy+tidx));
	}
	__syncthreads();
	sum_0=shared_0[tidx][((((nbidy*32)+tidy)+0)+(( - 1)*coalesced_idy))];
	sum_1=shared_0[tidx][((((nbidy*32)+tidy)+4)+(( - 1)*coalesced_idy))];
	sum_2=shared_0[tidx][((((nbidy*32)+tidy)+8)+(( - 1)*coalesced_idy))];
	sum_3=shared_0[tidx][((((nbidy*32)+tidy)+12)+(( - 1)*coalesced_idy))];
	sum_4=shared_0[tidx][((((nbidy*32)+tidy)+16)+(( - 1)*coalesced_idy))];
	sum_5=shared_0[tidx][((((nbidy*32)+tidy)+20)+(( - 1)*coalesced_idy))];
	sum_6=shared_0[tidx][((((nbidy*32)+tidy)+24)+(( - 1)*coalesced_idy))];
	sum_7=shared_0[tidx][((((nbidy*32)+tidy)+28)+(( - 1)*coalesced_idy))];
	__syncthreads();
	__syncthreads();
	C((((nbidy*32)+tidy)+0), (tidx+(blockDimX*nbidx)))=sum_0;
	C((((nbidy*32)+tidy)+4), (tidx+(blockDimX*nbidx)))=sum_1;
	C((((nbidy*32)+tidy)+8), (tidx+(blockDimX*nbidx)))=sum_2;
	C((((nbidy*32)+tidy)+12), (tidx+(blockDimX*nbidx)))=sum_3;
	C((((nbidy*32)+tidy)+16), (tidx+(blockDimX*nbidx)))=sum_4;
	C((((nbidy*32)+tidy)+20), (tidx+(blockDimX*nbidx)))=sum_5;
	C((((nbidy*32)+tidy)+24), (tidx+(blockDimX*nbidx)))=sum_6;
	C((((nbidy*32)+tidy)+28), (tidx+(blockDimX*nbidx)))=sum_7;
}
