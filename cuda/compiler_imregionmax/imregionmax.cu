// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <imregionmax_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C" void computeGold(float*, const float*, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
	runTest(argc, argv);

	CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) {
	CUT_DEVICE_INIT(argc, argv);

	// set seed for rand()
	srand(2006);

	// allocate host memory for matrices A
	unsigned int size_A = WIDTH_A * HEIGHT_A;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*) malloc(mem_size_A);

	// initialize host memory
	randomInit(h_A, size_A);

	for (int i = 0; i < WIDTH_A; i++) {
		for (int j = 0; j < HEIGHT_A; j++) {
			if (i < 15 || j < 15 || i > WIDTH_A - 2 || j > HEIGHT_A - 2) {
				h_A[j * WIDTH_A + i] = 0.0f;
			}
		}
	}

	// allocate device memory
	float* d_A;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));

	// copy host memory to device
	CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

	// allocate device memory for result
	unsigned int size_C = WIDTH_C * HEIGHT_C;
	unsigned int mem_size_C = sizeof(float) * size_C;

	// allocate host memory for the result
	float* h_C = (float*) malloc(mem_size_C);

	// create and start timer
	unsigned int timer = 0;

	// compute reference solution
	float* reference = (float*) malloc(mem_size_C);
	computeGold(reference, h_A, WIDTH_A, HEIGHT_A);
	CUTBoolean res;

	{
		free(h_C);
		h_C = (float*) malloc(mem_size_C);
		float* d_C;
		CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(16, 16);
		dim3 grid(WIDTH_C / threads.x, HEIGHT_C / threads.y);

		CUT_SAFE_CALL(cutCreateTimer(&timer));
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStartTimer(timer));
		// execute the kernel
		imregionmax_naive<<< grid, threads >>>(d_A, d_C, WIDTH_A);
		// stop and destroy timer
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStopTimer(timer));

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

		printf("imregionmax_naive Processing time: %f (ms), %f Gflops \n",
				cutGetTimerValue(timer), 2000.0 * 0 / cutGetTimerValue(timer)
						/ 1024 / 1024 / 1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
		CUDA_SAFE_CALL(hipFree(d_C));
	}
	// check result
	res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
	printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");


	{
		free(h_C);
		h_C = (float*) malloc(mem_size_C);
		float* d_C;
		CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(32, 1);
		dim3 grid(WIDTH_C / threads.x, WIDTH_C / (1));

		CUT_SAFE_CALL(cutCreateTimer(&timer));
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStartTimer(timer));
		// execute the kernel
		imregionmax_coalesced<<< grid, threads >>>(d_A, d_C, WIDTH_A);
		// stop and destroy timer
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStopTimer(timer));

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

		printf("imregionmax_coalesced Processing time: %f (ms), %f Gflops \n",
				cutGetTimerValue(timer), 2000.0 * 0 / cutGetTimerValue(timer)
						/ 1024 / 1024 / 1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
		CUDA_SAFE_CALL(hipFree(d_C));
	}
	// check result
	res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
	printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");



	{
		free(h_C);
		h_C = (float*) malloc(mem_size_C);
		float* d_C;
		CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(256, 1);
		dim3 grid(WIDTH_C / threads.x, WIDTH_C / (32));

		CUT_SAFE_CALL(cutCreateTimer(&timer));
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStartTimer(timer));
		// execute the kernel
		imregionmax_opt<<< grid, threads >>>(d_A, d_C, WIDTH_A);
		// stop and destroy timer
		hipDeviceSynchronize();
		CUT_SAFE_CALL(cutStopTimer(timer));

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

		printf("imregionmax_opt Processing time: %f (ms), %f Gflops \n",
				cutGetTimerValue(timer), 2000.0 * 0 / cutGetTimerValue(timer)
						/ 1024 / 1024 / 1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
		CUDA_SAFE_CALL(hipFree(d_C));
	}
	// check result
	res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
	printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");


	// clean up memory
	free(h_A);
	free(h_C);
	free(reference);
	CUDA_SAFE_CALL(hipFree(d_A));
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
	for (int i = 0; i < size; ++i)
		data[i] = rand() / (float) RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height) {
	int i, j, k;
	int error_count = 0;
	for (j = 0; j < height; j++) {
		for (i = 0; i < width; i++) {
			k = j * width + i;
			if (data1[k] != data2[k]) {
				printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i, j, data1[k],
						data2[k]);
				error_count++;
			}
		}
	}
	printf(" nTotal Errors = %d n", error_count);
}
