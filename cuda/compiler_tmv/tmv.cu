
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <tmv_kernel.cu>
#include <hipblas.h>
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    CUT_DEVICE_INIT(argc, argv);


    float result[1024];
    hipblasStatus_t status;
    status = cublasInit();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! CUBLAS initialization error\n");
	exit (1);
    }
    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

    // create and start timer
    unsigned int timer = 0;

    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);
    CUTBoolean res;

    {
        free(h_C);
        h_C = (float*) malloc(mem_size_C);
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(256, 1);
		dim3 grid(WC / threads.x, HC / threads.y);

		for (int i=0; i<16; i++) {
			CUT_SAFE_CALL(cutCreateTimer(&timer));
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStartTimer(timer));
			// execute the kernel
			tmv_naive<<< grid, threads >>>(d_A, d_B, d_C, WA);
			// stop and destroy timer
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStopTimer(timer));
		}

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
								  hipMemcpyDeviceToHost) );

		result[0] = cutGetTimerValue(timer);
		printf("tmv_naive Processing time: %f (ms), %f Gflops \n", cutGetTimerValue(timer), 2000.0*MW*MW/cutGetTimerValue(timer)/1024/1024/1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
	    CUDA_SAFE_CALL(hipFree(d_C));
    }
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");


    {
        free(h_C);
        h_C = (float*) malloc(mem_size_C);
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));

        for (int i=0; i<16; i++) {
			CUT_SAFE_CALL(cutCreateTimer(&timer));
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStartTimer(timer));
			// execute the kernel
			hipblasSgemv('n', MW, MW, 1.0f, d_A,
					MW, d_B, 1, 0.0f, d_C, 1);
			/*
			hipblasSgemm('n', 'n', MW, 1, MW, 1.0f, d_A,
					MW, d_B, MW, 0.0f, d_C, MW);*/
			// stop and destroy timer
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStopTimer(timer));
        }

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
								  hipMemcpyDeviceToHost) );

		printf("hipblasSgemm Processing time: %f (ms), %f Gflops \n", cutGetTimerValue(timer), 2000.0*MW*MW/cutGetTimerValue(timer)/1024/1024/1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
	    CUDA_SAFE_CALL(hipFree(d_C));

    }
    res = cutCompareL2fe(reference, h_C, size_C, 1e-4f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");




    {
        free(h_C);
        h_C = (float*) malloc(mem_size_C);
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(32, 1);
		dim3 grid(WC / threads.x, HC / threads.y);

		for (int i=0; i<16; i++) {
			CUT_SAFE_CALL(cutCreateTimer(&timer));
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStartTimer(timer));
			// execute the kernel
			tmv_coalesced<<< grid, threads >>>(d_A, d_B, d_C, WA);
			// stop and destroy timer
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStopTimer(timer));
		}

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
								  hipMemcpyDeviceToHost) );

		result[5] = cutGetTimerValue(timer);
		printf("tmv_coalesced Processing time: %f (ms), %f Gflops \n", cutGetTimerValue(timer), 2000.0*MW*MW/cutGetTimerValue(timer)/1024/1024/1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
	    CUDA_SAFE_CALL(hipFree(d_C));
    }
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");



    {
        free(h_C);
        h_C = (float*) malloc(mem_size_C);
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(512, 1);
		dim3 grid(WC / threads.x, HC / threads.y);

		for (int i=0; i<16; i++) {
			CUT_SAFE_CALL(cutCreateTimer(&timer));
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStartTimer(timer));
			// execute the kernel
			tmv_opt<<< grid, threads >>>(d_A, d_B, d_C, WA);
			// stop and destroy timer
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStopTimer(timer));
		}

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
								  hipMemcpyDeviceToHost) );

		result[5] = cutGetTimerValue(timer);
		printf("tmv_compiler Processing time: %f (ms), %f Gflops \n", cutGetTimerValue(timer), 2000.0*MW*MW/cutGetTimerValue(timer)/1024/1024/1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
	    CUDA_SAFE_CALL(hipFree(d_C));
    }
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");

    {
        free(h_C);
        h_C = (float*) malloc(mem_size_C);
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
		// setup execution parameters
		dim3 threads(512, 1);
		dim3 grid(WC / threads.x, HC / threads.y);

		for (int i=0; i<16; i++) {
			CUT_SAFE_CALL(cutCreateTimer(&timer));
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStartTimer(timer));
			// execute the kernel
			tmv_pref<<< grid, threads >>>(d_A, d_B, d_C, WA);
			// stop and destroy timer
			hipDeviceSynchronize();
			CUT_SAFE_CALL(cutStopTimer(timer));
		}

		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");

		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
								  hipMemcpyDeviceToHost) );

		result[5] = cutGetTimerValue(timer);
		printf("tmv_pref Processing time: %f (ms), %f Gflops \n", cutGetTimerValue(timer), 2000.0*MW*MW/cutGetTimerValue(timer)/1024/1024/1024);
		CUT_SAFE_CALL(cutDeleteTimer(timer));
	    CUDA_SAFE_CALL(hipFree(d_C));
    }
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");


    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    CUDA_SAFE_CALL(hipFree(d_A));
    CUDA_SAFE_CALL(hipFree(d_B));
    status = cublasShutdown();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! shutdown error\n");
    }
//    CUDA_SAFE_CALL(hipFree(d_C));
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

