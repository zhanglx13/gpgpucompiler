#include "hip/hip_runtime.h"

#ifndef _REDUCTION_KERNEL_H_
#define _REDUCTION_KERNEL_H_

#include <stdio.h>

#include "reduction.h"


__global__ void reduction_naive(float* d_odata, float* d_idata, int num_elements)
{

	int idx = (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	d_odata[idx] = d_idata[idx]+d_idata[idx+num_elements/2];

}

#define COALESCED_NUM 16
#define blockDimX 512
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimX 65536
#define globalDimY 1
__global__ void reduction_opt_0(float * A, int size, int segSize)
{
	#pragma	gCompiler	gValue	segSize	262144

	int k;
	float sum;
	int nidx;
	__shared__ float shared_0[512];
	nidx=((((tidx/16)*2048)+(idx&15))+((idx/512)*16));
	float tmp_2;
	float tmp_3;
	float tmp_0;
	float tmp_1;
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float r;
		r=A[(nidx+k)];
		sum+=r;
	}
	tmp_0=sum;
	__syncthreads();
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float r;
		r=A[((nidx+131072)+k)];
		sum+=r;
	}
	tmp_1=sum;
	__syncthreads();
	float a;
	float b;
	float c;
	a=tmp_0;
	b=tmp_1;
	c=(a+b);
	tmp_2=c;
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float r;
		r=A[((nidx+65536)+k)];
		sum+=r;
	}
	tmp_0=sum;
	__syncthreads();
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float r;
		r=A[(((nidx+65536)+131072)+k)];
		sum+=r;
	}
	tmp_1=sum;
	__syncthreads();
	a=tmp_0;
	b=tmp_1;
	c=(a+b);
	tmp_3=c;
	a=tmp_2;
	b=tmp_3;
	c=(a+b);
	shared_0[(tidx+0)]=c;
	__syncthreads();
	if ((nidx<32768))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+256)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<16384))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+128)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<8192))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+64)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<4096))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+32)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<2048))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+16)];
		c=(a+b);
		{
			A[nidx]=c;
		}
	}
}


#define COALESCED_NUM 16
#define blockDimX 512
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimX 512
#define globalDimY 1
__global__ void reduction_opt_1(float * A, int size, int segSize)
{
	#pragma	gCompiler	gValue	segSize	262144

	__shared__ float shared_1[512];
	float tmp_4;
	float tmp_5;
	float a;
	float b;
	float c;
	{
		a=A[idx];
	}
	{
		b=A[(idx+((262144/128)/2))];
	}
	c=(a+b);
	tmp_4=c;
	{
		a=A[(idx+512)];
	}
	{
		b=A[((idx+512)+((262144/128)/2))];
	}
	c=(a+b);
	tmp_5=c;
	a=tmp_4;
	b=tmp_5;
	c=(a+b);
	shared_1[(tidx+0)]=c;
	__syncthreads();
	if ((idx<256))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+256)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<128))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+128)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<64))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+64)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<32))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+32)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<16))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+16)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<8))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+8)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<4))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+4)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<2))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+2)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<1))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+1)];
		c=(a+b);
		{
			A[idx]=c;
		}
	}
}



#define COALESCED_NUM 16
#define blockDimX 512
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimX 65536
#define globalDimY 1
__global__ void reduction_complex_opt_0(float * A, float * B, int size, int segSize)
{
	#pragma	gCompiler	gValue	segSize	262144

	int k;
	float sum;
	int nidx;
	__shared__ float shared_0[512];
	nidx=((((tidx/16)*2048)+(idx&15))+((idx/512)*16));
	float tmp_4;
	float tmp_5;
	float tmp_2;
	float tmp_3;
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float real;
		float img;
		struct float2 * tmp_0;
		struct float2 tmp_1;
		tmp_0=((struct float2 * )A);
		tmp_1=tmp_0[(nidx+k)];
		real=tmp_1.x;
		img=tmp_1.y;
		sum+=real;
		sum+=img;
	}
	tmp_2=sum;
	__syncthreads();
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float real;
		float img;
		struct float2 * tmp_0;
		struct float2 tmp_1;
		tmp_0=((struct float2 * )A);
		tmp_1=tmp_0[((nidx+131072)+k)];
		real=tmp_1.x;
		img=tmp_1.y;
		sum+=real;
		sum+=img;
	}
	tmp_3=sum;
	__syncthreads();
	float a;
	float b;
	float c;
	a=tmp_2;
	b=tmp_3;
	c=(a+b);
	tmp_4=c;
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float real;
		float img;
		struct float2 * tmp_0;
		struct float2 tmp_1;
		tmp_0=((struct float2 * )A);
		tmp_1=tmp_0[((nidx+65536)+k)];
		real=tmp_1.x;
		img=tmp_1.y;
		sum+=real;
		sum+=img;
	}
	tmp_2=sum;
	__syncthreads();
	sum=0;
	for (k=0; k<size; k=(k+262144))
	{
		float real;
		float img;
		struct float2 * tmp_0;
		struct float2 tmp_1;
		tmp_0=((struct float2 * )A);
		tmp_1=tmp_0[(((nidx+65536)+131072)+k)];
		real=tmp_1.x;
		img=tmp_1.y;
		sum+=real;
		sum+=img;
	}
	tmp_3=sum;
	__syncthreads();
	a=tmp_2;
	b=tmp_3;
	c=(a+b);
	tmp_5=c;
	a=tmp_4;
	b=tmp_5;
	c=(a+b);
	shared_0[(tidx+0)]=c;
	__syncthreads();
	if ((nidx<32768))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+256)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<16384))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+128)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<8192))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+64)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<4096))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+32)];
		c=(a+b);
		shared_0[(tidx+0)]=c;
	}
	__syncthreads();
	if ((nidx<2048))
	{
		float a;
		float b;
		float c;
		a=shared_0[(tidx+0)];
		b=shared_0[(tidx+16)];
		c=(a+b);
		{
			B[nidx]=c;
		}
	}
}


#define COALESCED_NUM 16
#define blockDimX 512
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimX 512
#define globalDimY 1
__global__ void reduction_complex_opt_1(float * A, float * B, int size, int segSize)
{
	#pragma	gCompiler	gValue	segSize	262144

	__shared__ float shared_1[512];
	float tmp_6;
	float tmp_7;
	float a;
	float b;
	float c;
	{
		a=B[idx];
	}
	{
		b=B[(idx+((262144/128)/2))];
	}
	c=(a+b);
	tmp_6=c;
	{
		a=B[(idx+512)];
	}
	{
		b=B[((idx+512)+((262144/128)/2))];
	}
	c=(a+b);
	tmp_7=c;
	a=tmp_6;
	b=tmp_7;
	c=(a+b);
	shared_1[(tidx+0)]=c;
	__syncthreads();
	if ((idx<256))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+256)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<128))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+128)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<64))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+64)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<32))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+32)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<16))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+16)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<8))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+8)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<4))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+4)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<2))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+2)];
		c=(a+b);
		shared_1[(tidx+0)]=c;
	}
	__syncthreads();
	if ((idx<1))
	{
		float a;
		float b;
		float c;
		a=shared_1[(tidx+0)];
		b=shared_1[(tidx+1)];
		c=(a+b);
		{
			B[idx]=c;
		}
	}
}


#endif // #ifndef _REDUCTION_KERNEL_H_
