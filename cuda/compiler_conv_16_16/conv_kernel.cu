#include "hip/hip_runtime.h"

#ifndef _CONV_KERNEL_H_
#define _CONV_KERNEL_H_

#include <stdio.h>
#include "conv.h"

#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define blockDimX 16
#define blockDimY 16
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
__global__ void conv_naive(float *A, float *B, float *C, int width, int height, int w, int h) {
	int i;
	int j;
	float sum = 0;
	for (j=0; j<16; j=j+1) {
		for (i=0; i<16; i=i+1) {
			float a;
			float b;
			a = A(idy-j+h, idx-i+w);
			b = B(j, i);
			sum += a*b;
		}
	}
	C(idy, idx) = sum;
}




#define COALESCED_NUM 16
#define blockDimX 256
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 8
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
__global__ void conv_opt(float * A, float * B, float * C, int width, int height, int w, int h)
{
	__shared__ float shared_1[16][9];
	__shared__ float shared_0[272];
	int j;
	float sum_0 = 0;
	float sum_1 = 0;
	float sum_2 = 0;
	float sum_3 = 0;
	float sum_4 = 0;
	float sum_5 = 0;
	float sum_6 = 0;
	float sum_7 = 0;
	int it_2;
	for (j=0; j<(h-7); j=(j+1))
	{
		int it_2;
		if ((tidx<16))
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*j))+h), (idx+(( - 1)*0)));
		}
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*j))+h), ((idx+(( - 1)*0))+16));
		__syncthreads();
		if ((tidx<16))
		{
			shared_1[(tidx+0)][0]=B((j+0), (0+tidx));
			shared_1[(tidx+0)][1]=B((j+1), (0+tidx));
			shared_1[(tidx+0)][2]=B((j+2), (0+tidx));
			shared_1[(tidx+0)][3]=B((j+3), (0+tidx));
			shared_1[(tidx+0)][4]=B((j+4), (0+tidx));
			shared_1[(tidx+0)][5]=B((j+5), (0+tidx));
			shared_1[(tidx+0)][6]=B((j+6), (0+tidx));
			shared_1[(tidx+0)][7]=B((j+7), (0+tidx));
		}
		__syncthreads();
		#pragma unroll
		for (it_2=0; it_2<16; it_2=(it_2+1))
		{
			float a;
			float b_0;
			float b_1;
			float b_2;
			float b_3;
			float b_4;
			float b_5;
			float b_6;
			float b_7;
			a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
			b_0=shared_1[it_2][0];
			b_1=shared_1[it_2][1];
			b_2=shared_1[it_2][2];
			b_3=shared_1[it_2][3];
			b_4=shared_1[it_2][4];
			b_5=shared_1[it_2][5];
			b_6=shared_1[it_2][6];
			b_7=shared_1[it_2][7];
			sum_0+=(a*b_0);
			sum_1+=(a*b_1);
			sum_2+=(a*b_2);
			sum_3+=(a*b_3);
			sum_4+=(a*b_4);
			sum_5+=(a*b_5);
			sum_6+=(a*b_6);
			sum_7+=(a*b_7);
		}
		__syncthreads();
		__syncthreads();
	}
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-1)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-1)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		sum_0+=(a*b_0);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-2)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-2)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-3)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-3)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-4)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-4)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-5)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-5)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-6)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-6)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(h-7)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(h-7)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][0]=B((h-7), (0+tidx));
		}
		{
			shared_1[(tidx+0)][1]=B((h-6), (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B((h-5), (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B((h-4), (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B((h-3), (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B((h-2), (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B((h-1), (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_0;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_0=shared_1[it_2][0];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		sum_0+=(a*b_0);
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
	}
	{
		C(((idy*8)+0), idx)=sum_0;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-1)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-1)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][1]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][2]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(5, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(6, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_1;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_1=shared_1[it_2][1];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_1+=(a*b_1);
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+1), idx)=sum_1;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-2)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-2)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][2]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][3]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(4, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(5, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_2;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_2=shared_1[it_2][2];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_2+=(a*b_2);
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+2), idx)=sum_2;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-3)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-3)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][3]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][4]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(3, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(4, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_3;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_3=shared_1[it_2][3];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_3+=(a*b_3);
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+3), idx)=sum_3;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-4)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-4)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][4]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][5]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(2, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(3, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_4;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_4=shared_1[it_2][4];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_4+=(a*b_4);
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+4), idx)=sum_4;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-5)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-5)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][5]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][6]=B(1, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(2, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_5;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_5=shared_1[it_2][5];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_5+=(a*b_5);
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+5), idx)=sum_5;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-6)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-6)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][6]=B(0, (0+tidx));
		}
		{
			shared_1[(tidx+0)][7]=B(1, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_6;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_6=shared_1[it_2][6];
		b_7=shared_1[it_2][7];
		sum_6+=(a*b_6);
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+6), idx)=sum_6;
	}
	__syncthreads();
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-7)))+h), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-7)))+h), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_1[(tidx+0)][7]=B(0, (0+tidx));
		}
	}
	__syncthreads();
	#pragma unroll
	for (it_2=0; it_2<16; it_2=(it_2+1))
	{
		float a;
		float b_7;
		a=shared_0[((tidx+(( - 1)*(it_2+0)))+16)];
		b_7=shared_1[it_2][7];
		sum_7+=(a*b_7);
	}
	{
		C(((idy*8)+7), idx)=sum_7;
	}
	__syncthreads();
	__syncthreads();
}


#endif // #ifndef _CONV_KERNEL_H_
